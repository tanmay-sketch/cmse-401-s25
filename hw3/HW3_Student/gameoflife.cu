#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include "png_util.h"
#include <hip/hip_runtime.h>
#define MAX_N 20000
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) std::cout << "CUDA error: " << #x << " returned " << hipGetErrorString(cuda_error__) << std::endl;}

char plate[2][(MAX_N + 2) * (MAX_N + 2)];
int which = 0;
int n;

__global__ void iteration_kernel(char* d_plate, int n, int which) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    if(i <= n && j <= n) {
        int stride = n + 2;
        int index = i * stride + j;
        int plate_size = (n + 2) * (n + 2);
        char* curr = d_plate + which * plate_size;
        char* next = d_plate + ((which ^ 1) * plate_size);
        
        // Match the original live() function pattern
        int num = curr[index - stride - 3] 
                + curr[index - stride - 2]
                + curr[index - stride - 1]
                + curr[index - 1]
                + curr[index + 1]
                + curr[index + stride + 1]
                + curr[index + stride + 2]
                + curr[index + stride + 3];
        
        if(curr[index]) {
            next[index] = (num == 2 || num == 3) ? 1 : 0;
        } else {
            next[index] = (num == 3);
        }
    }
}

void print_plate(){
    if (n < 60) {
        for(int i = 1; i <= n; i++){
            for(int j = 1; j <= n; j++){
                printf("%d", (int) plate[which][i * (n + 2) + j]);
            }
            printf("\n");
        }
    } else {
        printf("Plate too large to print to screen\n");
    }
    printf("\0");
}

void plate2png(char* filename) {
    char* img = (char*) malloc(n*n*sizeof(char));
    image_size_t sz;
    sz.width = n;
    sz.height = n; 

    for(int i = 1; i <= n; i++){
        for(int j = 1; j <= n; j++){
            int pindex = i * (n + 2) + j;
            int index = (i-1) * (n) + j;
            if (plate[which][pindex] > 0)
                img[index] = 255; 
            else 
                img[index] = 0;
        }
    }
    printf("Writing file\n");
    write_png_file(filename, img, sz);
   
    printf("done writing png\n"); 
    free(img);
    printf("done freeing memory\n");
}

int main() {
    int M;
    char line[MAX_N];
    if(scanf("%d %d", &n, &M) == 2) {
        // Check for unreasonable size to prevent out-of-memory errors
        if (n <= 0) {
            n = 1000; // More reasonable default size
        } else if (n > 5000) {
            printf("Warning: Grid size %d is very large. Consider using a smaller size (<= 5000).\n", n);
        }
        
        memset(plate[0], 0, sizeof(char) * (n + 2) * (n + 2));
        memset(plate[1], 0, sizeof(char) * (n + 2) * (n + 2));
        
        if (n <= MAX_N && n > 0) {
            // Initialize from input
            for(int i = 1; i <= n; i++){
                scanf("%s", line);
                for(int j = 0; j < n; j++){
                    plate[0][i * (n + 2) + j + 1] = line[j] - '0';
                }
            }
        } else {
            // Random initialization
            for(int i = 1; i <= n; i++) 
                for(int j = 0; j < n; j++) 
                    plate[0][i * (n+2) + j + 1] = (char) rand() % 2;
        }

        // ------- GPU initialization ------- 
        int plate_size = (n + 2) * (n + 2) * sizeof(char);
        int total_size = 2 * plate_size;
        char* d_plate;

        // -------- Allocating and copying to GPU ---------
        CUDA_CALL(hipMalloc((void**)&d_plate, total_size));
        CUDA_CALL(hipMemcpy(d_plate, plate[0], total_size, hipMemcpyHostToDevice));
        
        dim3 block(16, 16); // defines a block of 16x16 threads
        dim3 grid((n + block.x - 1)/block.x, (n + block.y - 1)/block.y); // calculates how many blocks are needed
        
        for(int i = 0; i < M; i++) {
            printf("\nIteration %d:\n", i);
            if(n < 60) {
                CUDA_CALL(hipMemcpy(plate[which], d_plate + which * plate_size, plate_size, hipMemcpyDeviceToHost));
                print_plate();
            } else {
                printf("Plate too large to print to screen\n");
            }
            
            iteration_kernel<<<grid, block>>>(d_plate, n, which);
            CUDA_CALL(hipGetLastError()); // checks for any error
            CUDA_CALL(hipDeviceSynchronize()); // ensures kernel has completed its work
            which ^= 1; // switches the board using xor
        }
        
        // Copy the final state back to host
        CUDA_CALL(hipMemcpy(plate[which], d_plate + which * plate_size, plate_size, hipMemcpyDeviceToHost));
        
        printf("\n\nFinal:\n");
        plate2png("plate.png");
        print_plate();
        
        // Free GPU memory
        CUDA_CALL(hipFree(d_plate));
    }
    return 0;
}